#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "type.h"

#define MB_BACKOFF 1000


void gpu_init()
{
  hipSetDevice(0);
  hipDeviceReset();
}



int get_dims()
{
  size_t mem_free, mem_total;
  
  hipMemGetInfo(&mem_free, &mem_total);
  return (int) sqrt((mem_total - MB_BACKOFF*1024*1024)/sizeof(REAL));
}



void gpu_to_host(void *x_cpu, void *x_gpu, size_t len)
{
  hipMemcpy(x_cpu, x_gpu, len, hipMemcpyDeviceToHost);
}



void gpu_free(void *x)
{
  if (x)
    hipFree(x);
}
