#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "type.h"

#define MB_BACKOFF 1000


void gpu_init()
{
  hipSetDevice(0);
  hipDeviceReset();
}



int get_dims()
{
  size_t mem_free, mem_total;
  
  hipMemGetInfo(&mem_free, &mem_total);
  
  double bytes = (double)(mem_total - (size_t)MB_BACKOFF*1024*1024);
  return (int) sqrt(bytes/sizeof(REAL));
}



void gpu_to_host(void *x_cpu, void *x_gpu, size_t len)
{
  hipMemcpy(x_cpu, x_gpu, len, hipMemcpyDeviceToHost);
}



void gpu_free(void *x)
{
  if (x)
    hipFree(x);
}
