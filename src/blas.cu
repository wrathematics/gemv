#include <hipblas.h>

#include "common.h"
#include "restrict.h"
#include "type.h"

hipblasOperation_t trans;
int k;
int inc;
hipblasHandle_t handle;
REAL *alpha;
REAL *beta;


int mvm_init()
{
  trans = HIPBLAS_OP_N;
  k = 1;
  inc = 1;
  
  const REAL alpha_cpu = (REAL) 1.0;
  const REAL beta_cpu = (REAL) 0.0;
  hipMalloc(&alpha, sizeof(*alpha));
  hipMalloc(&beta, sizeof(*beta));
  if (alpha == NULL || beta == NULL)
    return ERR_CUMALLOC;
  
  hipMemcpy(alpha, &alpha_cpu, 1, hipMemcpyHostToDevice);
  hipMemcpy(beta, &beta_cpu, 1, hipMemcpyHostToDevice);
  
  hipblasStatus_t st = hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  
  if (st != HIPBLAS_STATUS_SUCCESS)
    return ERR_CUBLAS;
  else
    return ERR_OK;
}



void mvm_cleanup()
{
  hipFree(alpha);
  hipFree(beta);
  hipblasDestroy(handle);
}



// ----------------------------------------------------------------------------
// c    = A    *    b
//  mx1    mxn       nx1
// ----------------------------------------------------------------------------

void mvm_gemm(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
#if TYPE == FLOAT
  hipblasSgemm(handle, trans, trans, m, k, n, alpha, A, m, b, n, beta, c, m);
#elif TYPE == DOUBLE
  hipblasDgemm(handle, trans, trans, m, k, n, alpha, A, m, b, n, beta, c, m);
#endif
}



void mvm_gemv(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
#if TYPE == FLOAT
  hipblasSgemv(handle, trans, m, n, alpha, A, m, b, inc, beta, c, inc);
#elif TYPE == DOUBLE
  hipblasDgemv(handle, trans, m, n, alpha, A, m, b, inc, beta, c, inc);
#endif
}
