#include <hipblas.h>

#include "common.h"
#include "restrict.h"
#include "type.h"

hipblasHandle_t handle;

int mvm_init()
{
  hipblasStatus_t st = hipblasCreate(&handle);
  if (st != HIPBLAS_STATUS_SUCCESS)
    return ERR_CUBLAS;
  else
    return ERR_OK;
}



void mvm_cleanup()
{
  hipblasDestroy(handle);
}



// ----------------------------------------------------------------------------
// c    = A    *    b
//  mx1    mxn       nx1
// ----------------------------------------------------------------------------

void mvm_gemm(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
  const hipblasOperation_t trans = HIPBLAS_OP_N;
  const REAL alpha = 1.0;
  const REAL beta = 0.0;
  const int k = 1;
  
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
#if TYPE == FLOAT
  hipblasSgemm(handle, trans, trans, m, k, n, &alpha, A, m, b, n, &beta, c, m);
#elif TYPE == DOUBLE
  hipblasDgemm(handle, trans, trans, m, k, n, &alpha, A, m, b, n, &beta, c, m);
#endif
}



void mvm_gemv(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
  const hipblasOperation_t trans = HIPBLAS_OP_N;
  const REAL alpha = 1.0;
  const REAL beta = 0.0;
  const int inc = 1;
  
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
#if TYPE == FLOAT
  hipblasSgemv(handle, trans, m, n, &alpha, A, m, b, inc, &beta, c, inc);
#elif TYPE == DOUBLE
  hipblasDgemv(handle, trans, m, n, &alpha, A, m, b, inc, &beta, c, inc);
#endif
}
