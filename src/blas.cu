#include <hipblas.h>

#include "common.h"
#include "restrict.h"
#include "type.h"

// ----------------------------------------------------------------------------
// c    = A    *    b
//  mx1    mxn       nx1
// ----------------------------------------------------------------------------

int mvm_gemm(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
  const hipblasOperation_t trans = HIPBLAS_OP_N;
  const REAL alpha = 1.0;
  const REAL beta = 0.0;
  const int k = 1;
  
  hipblasHandle_t handle;
  hipblasStatus_t st = hipblasCreate(&handle);
  if (st != HIPBLAS_STATUS_SUCCESS)
    return ERR_CUBLAS;
  
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
#if TYPE == FLOAT
  hipblasSgemm(handle, trans, trans, m, k, n, &alpha, A, m, b, n, &beta, c, m);
#elif TYPE == DOUBLE
  hipblasDgemm(handle, trans, trans, m, k, n, &alpha, A, m, b, n, &beta, c, m);
#endif
  hipblasDestroy(handle);
  
  return ERR_OK;
}



int mvm_gemv(const int m, const int n, const REAL *const restrict A,
  const REAL *const restrict b, REAL *const restrict c)
{
  const hipblasOperation_t trans = HIPBLAS_OP_N;
  const REAL alpha = 1.0;
  const REAL beta = 0.0;
  const int inc = 1;
  
  hipblasHandle_t handle;
  hipblasStatus_t st = hipblasCreate(&handle);
  if (st != HIPBLAS_STATUS_SUCCESS)
    return ERR_CUBLAS;
  
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
#if TYPE == FLOAT
  hipblasSgemv(handle, trans, m, n, &alpha, A, m, b, inc, &beta, c, inc);
#elif TYPE == DOUBLE
  hipblasDgemv(handle, trans, m, n, &alpha, A, m, b, inc, &beta, c, inc);
#endif
  hipblasDestroy(handle);
  
  return ERR_OK;
}
